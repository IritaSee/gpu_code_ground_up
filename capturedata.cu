#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <iostream>

#include "modules/globals.hpp"
#include "modules/commons.hpp"

char buffer[255];
drug_t ic50;
// __shared__ drug_t *d_ic50;
double *d_concs;

int get_IC50_data_from_file(const char* file_name, double *ic50)
{
  FILE *fp_drugs;
//   drug_t ic50;
  char *token;
  
  unsigned short idx;

  if( (fp_drugs = fopen(file_name, "r")) == NULL){
    printf("Cannot open file %s\n",
      file_name);
    return 0;
  }
  idx = 0;
  int sample_size = 0;
  fgets(buffer, sizeof(buffer), fp_drugs); // skip header
  while( fgets(buffer, sizeof(buffer), fp_drugs) != NULL )
  { // begin line reading
    token = strtok( buffer, "," );
    while( token != NULL )
    { // begin data tokenizing
      ic50[idx++] = strtod(token, NULL);
      token = strtok(NULL, ",");
    } // end data tokenizing
    sample_size++;
  } // end line reading

  fclose(fp_drugs);
  return sample_size;
}

__global__ void check(drug_t d_ic50, int sample_size){
 unsigned short sample_id;
    sample_id = threadIdx.x;
    { // begin sample loop
        printf("Sample_ID:%d \nData: ",
        sample_id );
        
        for (int z=0+(sample_id*14);z<(sample_id*14)+14;z++){
            printf("Core %d ic50[%d]: %lf \n",sample_id, z, d_ic50[z]);
        }
        // printf("\n");

    } // end sample loop
}

double concs[4];
int main()
{
    double *d_ic50;
    // input variables for cell simulation

    unsigned short idx;

    snprintf(buffer, sizeof(buffer),
      "./IC50_samples10.csv");
    int sample_size = get_IC50_data_from_file(buffer, ic50);
    // if(ic50.size() == 0)
    //     printf("Something problem with the IC50 file!\n");
    // else if(ic50.size() > 2000)
    //     printf("Too much input! Maximum sample data is 2000!\n");
    hipMalloc(&d_ic50, sizeof(drug_t));
    hipMemcpy(d_ic50, ic50, sizeof(drug_t), hipMemcpyHostToDevice);
    check<<<1,10>>>(d_ic50, sample_size);
    hipDeviceSynchronize();
    // unsigned short sample_id;
    // for( sample_id = 0;
    //     sample_id < sample_size;
    //     sample_id ++ )
    // { // begin sample loop
    //     printf("Sample_ID:%d \nData: ",
    //     sample_id );
        
    //     for (int z=0+(sample_id*14);z<(sample_id*14)+14;z++){
    //         printf("%lf ",ic50[z]);
    //     }
    //     printf("\n");

    // } // end sample loop
    
    return 0;
}
