#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <kat/

__global__ void hello_world() {
  kat::snprintf("Hello, world from CUDA!\n");
}

int main() {
  hello_world<<<1, 1>>>();
  hipDeviceSynchronize();

  return 0;
}