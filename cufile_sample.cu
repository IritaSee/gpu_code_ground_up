#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cufile.h>

#define CUDA_CHECK(status) \
    do { \
        hipError_t err = status; \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA error: %s at line %d\n", hipGetErrorString(err), __LINE__); \
            exit(1); \
        } \
    } while (0)

#define CUFILE_CHECK(status) \
    do { \
        cufile_error_t err = status; \
        if (err != CUFILE_SUCCESS) { \
            fprintf(stderr, "cuFile error: %s at line %d\n", cufile_error_string(err), __LINE__); \
            exit(1); \
        } \
    } while (0)

int main() {
    // Allocate and initialize data on the GPU
    const int numElements = 1024;
    const int bufferSize = numElements * sizeof(float);
    float* d_data;
    CUDA_CHECK(hipMalloc((void**)&d_data, bufferSize));
    CUDA_CHECK(hipMemset(d_data, 0, bufferSize));

    // Open the output file using cuFile
    const char* filePath = "output_file.bin";
    cufile_handle_t handle;
    CUFILE_CHECK(cuFileDriverOpen(&handle, CUFILE_DRIVER_GDS));

    // Write data from GPU to the file using cuFile
    const off_t offset = 0;
    CUFILE_CHECK(cuFileWrite(handle, d_data, bufferSize, offset, 0));

    // Close the file handle
    CUFILE_CHECK(cuFileDriverClose(handle));

    // Free allocated memory on the GPU
    CUDA_CHECK(hipFree(d_data));

    return 0;
}