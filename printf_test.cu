#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <kat/on_device/c_standard_library/printf.cuh>

__global__ void hello_world() {
  char buffer [100];
  kat::sprintf(buffer, ", and the half of that is %d.", 60/2/2 );
  //puts(buffer);
}

int main() {
  hello_world<<<1, 1>>>();
  hipDeviceSynchronize();

  return 0;
}
